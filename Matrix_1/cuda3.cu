#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#define BLOCK_SIZE 32
#define N 43008
// This is the kernal
__global__ void scanKernal(float* d_I, float* d_O) {
    // The ID for the thread selected (Including block dimensions).
    int globalid = blockDim.x * blockIdx.x + threadIdx.x;
    // The thread ID
    int threadid = threadIdx.x;
    // Creating a shared memory space
    int block = blockDim.x * 1;
    __shared__ float* temp;
    // printf("wow %i", block);
    __syncthreads();

    temp = new float[block];
    // printf("BSize: %i", block);
    __syncthreads();

    // Store the thread id and its value in shared memory. This is because accessing shared memory has a lower latency than global memory.
    temp[threadid] = d_I[globalid];
    // Waiting for threads to complete. [Thread guard]
    __syncthreads();
    printf("BLOCK DIM: %i", blockDim.x);

    for (int offset = 1; offset < N; offset *= 2) {
        if (threadid >= offset)

            temp[threadid] += temp[threadid - offset];
        __syncthreads();
    }
    d_O[globalid] = temp[threadid];
    printf("KOUT: %f", temp[threadid]);
}

__global__ void finalScan(float* d_firstScan, float* d_secondScan) {
    //printf("\n FINAL SCAN");
    // The ID for the thread selected (Including block dimensions).
    int globalid = blockDim.x * blockIdx.x + threadIdx.x;
    // The thread ID
    int threadid = threadIdx.x;
    // Creating a shared memory space
    int block = blockDim.x * 1;
    int block_id = blockIdx.x * 1;
    // This shared array stores the second scan. This is will be used to be added onto the final output.
    //__shared__ float* addifier;
    // The size of the array is N/BLOCK_SIZE as it is the number of blocks in the grid.
    //addifier = new float[N / BLOCK_SIZE];
    // Synchronising the threads.
    //__syncthreads();
    // Checking if the block id is greater than 0 because we don't want to change the first block values. Also, we don't want to go into negative values if we subtract one from the index.
    //if (blockIdx.x > 0) addifier[blockIdx.x - 1] = d_secondScan[blockIdx.x - 1];
    // Synchronising the threads.
    //__syncthreads();
    // Shared array that stores the first scan results.
    //__shared__ float* temp;
    //temp = new float[512];
    //__syncthreads();

    // Store the thread id and its value in shared memory. This is because accessing shared memory has a lower latency than global memory.
    //temp[threadid] = d_firstScan[globalid];
    // Waiting for threads to complete. [Thread guard]
    //__syncthreads();
    printf("BLOCK DIM: %i", blockDim.x);
    // Checking if the block ID is greater than zero as block ID zero must not be modified.

    if (blockIdx.x > 0) {
        // Add the second scan result onto the first scan result.

        d_firstScan[globalid] = d_firstScan[globalid] + d_secondScan[blockIdx.x - 1];;
        printf("\n Hmm %f \n", d_secondScan[blockIdx.x - 1]);

        // Synchornise the threads
        __syncthreads();
    }
    // Write back into global memory
    d_firstScan[globalid] = d_firstScan[globalid];
    // printf("\n KERN: %f \n ", temp[threadid]);
    __syncthreads();

}

float* addScan(float* x, float* y, int n, float * d_input, float * d_output) {
    float* g = new float[N];
    dim3 noBlocks;
    dim3 noThreads;
    noBlocks = dim3(n / BLOCK_SIZE);
    noThreads = dim3(BLOCK_SIZE);
    hipError_t err;
    //float* d_input, * d_aux;
    //err = hipMalloc((void**)&d_input, sizeof(float) * n);
    //printf("\nError:d_input %s\n", hipGetErrorString(err));
    //err = hipMalloc((void**)&d_output, sizeof(float) * n / BLOCK_SIZE);
    //printf("\nError:d_aux %s\n", hipGetErrorString(err));


    err = hipMemcpy(d_input, x, sizeof(float) * n, hipMemcpyHostToDevice);
    printf("\nError:hipMemcpy %s\n", hipGetErrorString(err));
    err = hipMemcpy(d_output, y, sizeof(float) * n / BLOCK_SIZE, hipMemcpyHostToDevice);
    printf("\nError:d_aux %s\n", hipGetErrorString(err));

    finalScan << < noBlocks, noThreads >> > (d_input, d_output);
    err = hipDeviceSynchronize();
    printf("\nError: hipDeviceSynchronize%s\n", hipGetErrorString(err));
    err = hipMemcpy(g, d_input, sizeof(float) * n, hipMemcpyDeviceToHost);
    printf("\nError: hipMemcpy %s\n", hipGetErrorString(err));

    //hipFree(d_input);
    //hipFree(d_output);

    return g;
}
float* getEnds(float* x, int n) {
    int n2 = n / BLOCK_SIZE;
    float* output = new float[n2];
    for (int i = 0; i < n / BLOCK_SIZE; i++) {
        output[i] = x[((i + 1) * BLOCK_SIZE) - 1];
    }
    return output;
}
float* runScan(float* x, int n, float * d_input, float * d_output) {
    dim3 noBlocks;
    dim3 noThreads;
    noBlocks = dim3(n / BLOCK_SIZE);
    noThreads = dim3(BLOCK_SIZE);
    //float* d_input, float* d_output;
    //hipMalloc((void**)&d_input, sizeof(float) * n);
    //hipMalloc((void**)&d_output, sizeof(float) * n);
    hipMemcpy(d_input, x, sizeof(float) * n, hipMemcpyHostToDevice);
    scanKernal << < noBlocks, noThreads >> > (d_input, d_output);
    hipDeviceSynchronize();
    hipMemcpy(x, d_output, sizeof(float) * n, hipMemcpyDeviceToHost);
    //hipFree(d_input);
    //hipFree(d_output);
    return x;

}
float* recursion(int n, float* aux) {
    //float* res = runScan(aux, n);
    return nullptr;
}

float* f(float* a, int n) {
    float placeholder = 0;
    for (int i = 0; i < n; i++) {
        a[i] = a[i] + placeholder;
        if ((i + 1) % BLOCK_SIZE == 0) {
            placeholder = a[i];
            //printf("F %f \n", a[i]);
        }
    }
    return a;
}
void scanMiddle(float* h_input, float* h_output) {
    float* d_input, * d_output;
    hipMalloc((void**)&d_input, sizeof(float) * N);
    hipMalloc((void**)&d_output, sizeof(float) * N);

    float* firstScan = runScan(h_input, N, d_input, d_output);
    float* ends = getEnds(firstScan, N);
    float* scanEnds = runScan(ends, N / BLOCK_SIZE, d_input, d_output);
    float* g = f(scanEnds, N / BLOCK_SIZE);
    float* addScans = addScan(firstScan, g, N, d_input, d_output);


    for (int i = 0; i < N; i++) {
        printf("\n %i END_SCAN %f", i, addScans[i]);
    }

    hipFree(d_input);
    hipFree(d_output);

}

int main() {
    // Creating an output and input array. Output array is the size of the grid. Input array is the number of elements we want to reduce.
    float* output = new float[N];
    float* input = new float[N];
    // This variable is just to check the real final answer
    float x = 0;
    // Looping through int he input array and placing the values we want to reduce.
    for (int i = 0; i < N; i++) {
        input[i] = 1.0f;
        // Adding the values to the variable x to see the real final answer computed by the CPU
        x = x + 1.0f;
    }
    // Outputting the real final answer.
    printf("\nReal Answer: %f\n", x);
    // Calling the middle function that calls the kernal.
    scanMiddle(input, output);
    hipDeviceReset();


    return 0;

}